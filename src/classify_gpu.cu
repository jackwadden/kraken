
#include <hip/hip_runtime.h>
#include "stdio.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
        {
            fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
            if (abort) exit(code);
        }
}

__global__ void vector_add(int *a, int *b, int length)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(id < length)
        a[id] = a[id] + b[id] + 5;
}

void kernel_wrapper(int *a, int *b, int length)
{

    int *a_d;
    int *b_d;

    int blockSize = 32;
    int numBlocks = (int)(length / blockSize);

    if(length % blockSize) {
        numBlocks++;
    }

    dim3 threads( blockSize, 1 );
    dim3 blocks( numBlocks, 1 );

    size_t byteLength = length * sizeof(int);

    gpuErrchk(hipMalloc( (void **)&a_d, byteLength ));
    gpuErrchk(hipMalloc( (void **)&b_d, byteLength ));

    gpuErrchk( hipMemcpy( a_d, a, byteLength, hipMemcpyHostToDevice ));
    gpuErrchk( hipMemcpy( b_d, b, byteLength, hipMemcpyHostToDevice ));

    vector_add<<< blocks, threads >>>( a_d, b_d , length);
    gpuErrchk( hipPeekAtLastError() );

    gpuErrchk( hipMemcpy( a, a_d, byteLength, hipMemcpyDeviceToHost ));
    gpuErrchk( hipMemcpy( b, b_d, byteLength, hipMemcpyDeviceToHost ));

    gpuErrchk( hipFree(a_d) );
    gpuErrchk( hipFree(b_d) );

}
